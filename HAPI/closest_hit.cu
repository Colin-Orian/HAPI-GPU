#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <stdio.h>
struct Payload {
	float3 colour;
	double diff;
	double t;
};

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(double, diff, attribute diff, ); //Where on the object does the ray intersect?
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(double, colour, attribute colour, );
//On the closest hit, make the pixel red
RT_PROGRAM void closestHit() {
	double ambient = 0.2;
	double diffuse = 0.8;
	double result = diff * diffuse * colour + ambient;


	payload.colour = make_float3(result);
}