#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
struct Payload {
	float3 colour;
	double diff;
	double t;
};

rtDeclareVariable(Payload, payload, rtPayload, );
//On a miss, make the pixel green
RT_PROGRAM void miss() {
	payload.colour += make_float3(0.0f, 0.0f, 0.0f);
}